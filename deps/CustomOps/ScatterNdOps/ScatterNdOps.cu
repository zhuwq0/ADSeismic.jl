#include "hip/hip_runtime.h"

__global__ void ScatterNdOps_forward_kernel(double *out, const long long*ii, const double *update, int n){
    int p =  blockIdx.x *blockDim.x + threadIdx.x;
    if (p<n){
        out[ii[p]-1] = update[p];
    }
}

void Gpu_ScatterNdOps_forward(double *out, const long long *ii,
    const double *update, int n){
    ScatterNdOps_forward_kernel<<< (n - 1)/64 + 1, 64 >>>(out, ii, update, n);
 }

 
 __global__ void ScatterNdOps_backward_kernel(double *grad_update, 
    const double *grad_out,
    const double *out, const long long *ii,
    const double *update, int n){
    int p =  blockIdx.x *blockDim.x + threadIdx.x;
    if (p<n) {
        grad_update[p] = grad_out[ii[p]-1];
    }
}

void Gpu_ScatterNdOps_backward(
    double *grad_update, 
    const double *grad_out,
    const double *out, const long long *ii,
    const double *update, int n){
    ScatterNdOps_backward_kernel<<< (n - 1)/64 + 1, 64 >>>(grad_update, grad_out, out, ii, update, n);
 }

 void get_ScatterNdOps_num(long long *out, const long long *m){
    hipMemcpy(out, m, sizeof(long long), hipMemcpyDeviceToHost);
 }